#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "kernels.h"
#include "../Scene/Tree.h"

#include <stdio.h>

__global__ void kernSpaceCol(Bud* dev_buds, const int numBuds, AttractorPoint* dev_attrPts, const int numAttractorPoints) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= numBuds) {
        return;
    }

}

hipError_t RunSpaceColonizationKernel(Bud* buds, const int numBuds, AttractorPoint* attractorPoints, const int numAttractorPoints) {
    hipError_t cudaStatus;

    Bud* dev_buds = 0;
    AttractorPoint* dev_attrPts = 0;

    // Device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Cuda Malloc
    cudaStatus = hipMalloc((void**)&dev_buds, numBuds * sizeof(Bud));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_attrPts, numAttractorPoints * sizeof(AttractorPoint));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Cuda memcpy
    cudaStatus = hipMemcpy(dev_buds, buds, numBuds * sizeof(Bud), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_attrPts, attractorPoints, numAttractorPoints * sizeof(AttractorPoint), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Run the kernel
    const int blockSize = 32;
    kernSpaceCol << < (numBuds + blockSize - 1) / blockSize, blockSize >> > (dev_buds, numBuds, dev_attrPts, numAttractorPoints);

    // Cuda Memcpy the Bud info back to the CPU
    cudaStatus = hipMemcpy(buds, dev_buds, numBuds * sizeof(Bud), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_buds);
    hipFree(dev_attrPts);

    return cudaStatus;
}

void TreeApp::PerformSpaceColonizationParallel(Bud* buds, const int numBuds, AttractorPoint* attractorPoints, const int numAttractorPoints) {
    hipError_t cudaStatus = RunSpaceColonizationKernel(buds, numBuds, attractorPoints, numAttractorPoints);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
    }
}
