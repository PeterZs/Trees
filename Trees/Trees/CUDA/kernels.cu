#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "kernels.h"
#include "../Scene/Tree.h"

#include <stdio.h>

__global__ void kernSpaceCol(Bud* dev_buds, const int numBuds, AttractorPoint* dev_attrPts, const int numAttractorPoints) {
    int index = threadIdx.x + (blockIdx.x * blockDim.x);
    if (index >= numBuds) {
        return;
    }

    Bud& currentBud = dev_buds[index];

    // Space Colonization
    if (currentBud.internodeLength > 0.0f && currentBud.fate == DORMANT) {
        for (int ap = 0; ap < numAttractorPoints; ++ap) {
            AttractorPoint& currentAttrPt = dev_attrPts[ap];
            glm::vec3 budToPtDir = currentAttrPt.point - currentBud.point; // Use current axillary or terminal bud
            const float budToPtDist2 = glm::length2(budToPtDir);
            budToPtDir = glm::normalize(budToPtDir);
            const float dotProd = glm::dot(budToPtDir, currentBud.naturalGrowthDir);
            if (budToPtDist2 < (12.0f * currentBud.internodeLength * currentBud.internodeLength) && dotProd > std::abs(COS_THETA_SMALL)) {
                if (budToPtDist2 < currentAttrPt.nearestBudDist2) {
                    currentAttrPt.nearestBudDist2 = budToPtDist2;
                    if (currentAttrPt.nearestBudBranchIdx != -1 && currentAttrPt.nearestBudIdx != -1) {
                        Bud& oldNearestBud = branches[attrPtIter->nearestBudBranchIdx].buds[attrPtIter->nearestBudIdx];
                        glm::vec3& oldNearestBudDir = oldNearestBud.optimalGrowthDir * (float)oldNearestBud.numNearbyAttrPts;
                        oldNearestBudDir -= budToPtDir;
                        if (--oldNearestBud.numNearbyAttrPts > 0) {
                            oldNearestBudDir = glm::normalize(oldNearestBudDir);
                        } else {
                            oldNearestBudDir = glm::vec3(0.0f);
                        }
                    }
                    currentAttrPt.nearestBudBranchIdx = ;
                    currentAttrPt.nearestBudIdx = ;
                    currentBud.optimalGrowthDir += budToPtDir;
                    ++currentBud.numNearbyAttrPts;
                }
            }
        }
    }

    if (currentBud.numNearbyAttrPts > 0) {
        currentBud.optimalGrowthDir = glm::normalize(currentBud.optimalGrowthDir);
        currentBud.environmentQuality = 1.0f;
    }
}

hipError_t RunSpaceColonizationKernel(Bud* buds, const int numBuds, AttractorPoint* attractorPoints, const int numAttractorPoints) {
    hipError_t cudaStatus;

    Bud* dev_buds = 0;
    AttractorPoint* dev_attrPts = 0;

    // Device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Cuda Malloc
    cudaStatus = hipMalloc((void**)&dev_buds, numBuds * sizeof(Bud));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_attrPts, numAttractorPoints * sizeof(AttractorPoint));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Cuda memcpy
    cudaStatus = hipMemcpy(dev_buds, buds, numBuds * sizeof(Bud), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_attrPts, attractorPoints, numAttractorPoints * sizeof(AttractorPoint), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Run the kernel
    const int blockSize = 32;
    kernSpaceCol << < (numBuds + blockSize - 1) / blockSize, blockSize >> > (dev_buds, numBuds, dev_attrPts, numAttractorPoints);

    // Cuda Memcpy the Bud info back to the CPU
    cudaStatus = hipMemcpy(buds, dev_buds, numBuds * sizeof(Bud), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_buds);
    hipFree(dev_attrPts);

    return cudaStatus;
}

void TreeApp::PerformSpaceColonizationParallel(Bud* buds, const int numBuds, AttractorPoint* attractorPoints, const int numAttractorPoints) {
    hipError_t cudaStatus = RunSpaceColonizationKernel(buds, numBuds, attractorPoints, numAttractorPoints);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
    }
}
